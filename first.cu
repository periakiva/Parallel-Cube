#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>



//define the multithread action
__global__ void cube(float * d_out, float * d_in){
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f*f*f;
}

//start main activity
int main(int argc,char **argv){

  //initilize array specs
    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  //initalize array values
    float h_in[ARRAY_SIZE];
    for(int i=0;i<ARRAY_SIZE;i++){
        h_in[i]=float(i);
    }

  //print array
    std::cout<<"Before: \n";
    for(int i=0;i<ARRAY_SIZE;i++){
        printf("%f", h_in[i]);
        printf(((i%4)!=3) ? "\t" : "\n");
  }

    std::cout<<"\n";

  //initlize an array of the same size as our input
    float h_out[ARRAY_SIZE];

  //initalize the inputs to the multithread functiuon
    float * d_in;
    float * d_out;

  //allocate memory for the arrays
    hipMalloc((void**) &d_in,ARRAY_BYTES);
    hipMalloc((void**) &d_out,ARRAY_BYTES);

  //error check
  //std::cout<<cudaGetErrorString(cudaGetLastError())<<std::endl;

  //copy array from CPU to GPU to preform function on GPU's threads
    hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);
    cube<<<1,ARRAY_SIZE>>>(d_out,d_in);
  //copy result from function back from GPU to CPU 
    hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);

  //print result array
    std::cout<<"\nAfter: \n";
    for(int i=0;i<ARRAY_SIZE;i++){
    printf("%f", h_out[i]);
    printf(((i%4)!=3) ? "\t" : "\n");
  }

    //free memory
    hipFree(d_in);
    hipFree(d_out);

  return 0;
}
